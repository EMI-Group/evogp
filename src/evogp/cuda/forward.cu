#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
// #include <thrust/execution_policy.h>
// #include <thrust/reduce.h>

// #undef CONST

// __constant__ GPNode<double> _constGP[MAX_STACK];

template<bool multiOutput = false>
__device__ inline void _treeGPEvalByStack(
    const float* value,
    const int16_t* type,
    const int16_t* subtree_size,
    const float* i_vars, // variables
    float* s_vals, // stack, size = MAX_STACK_SIZE
    int16_t* s_infos, // infos, size = 2 * MAX_STACK_SIZE
    const unsigned int n, 
    const unsigned int popSize, 
    const unsigned int maxGPLen, 
    const unsigned int varLen, 
    const unsigned int outLen, 
    float*& s_outs, 
    int& top
)
{
	/*
		s_vals: float*, stack memory stores the stack for operants. Also used to store tree values (avoid random access to global memory). length MAX_STACK_SIZE	
		s_infos: int16_t*, stack memory stores some useful infos. length 2 * MAX_STACK_SIZE
			SingleOutput: [0, MAX_STACK_SIZE): tree types(int16_t), [MAX_STACK_SIZE, 2 * MAX_STACK_SIZE): variable values (float)
			MultiOutput: [0, MAX_
			STACK_SIZE): tree types(int16_t), [MAX_STACK_SIZE, 1.5 * MAX_STACK_SIZE): variable values (float), [1.5 * MAX_STACK_SIZE, 2 * MAX_STACK_SIZE): output values (float)
	*/
	float* s_vars = (float*)(s_infos + MAX_STACK);  // variable values on stack
 	if constexpr (multiOutput)  // outLen > 0, otherwise outLen = 0
	{
		s_outs = (float*)(s_infos + MAX_STACK + MAX_STACK / 2);
        for (int i = 0; i < outLen; i++)
        {
            s_outs[i] = 0;  // output values on stack
        }
	}

	// load variable values from global memory to stack memory
	for (int i = 0; i < varLen; i++)
	{
		s_vars[i] = i_vars[i];
	}
    
	const unsigned int len = subtree_size[0];
    // load tree from global memory to stack memory
	// the order is: the inverse of prefix expression
	for (int i = 0; i < len; i++)
	{
		s_vals[len - 1 - i] = value[i];
		s_infos[len - 1 - i] = type[i];
	}

	// do stack operation according to the type of each node
	top = 0;
	for (int i = 0; i < len; i++)
	{
        // check node type
		int16_t node_type = s_infos[i];
		float node_value = s_vals[i];

		// for multiOutput
		int16_t is_outNode = 0;
		float right_node = 0;  

		if constexpr (multiOutput)
		{
			is_outNode = node_type & (int16_t)NodeType::OUT_NODE;
			node_type &= NodeType::TYPE_MASK;
		}

		// if the node is leaf
		if (node_type == NodeType::CONST)
		{
			s_vals[top++] = node_value;
			continue;
		}
		else if (node_type == NodeType::VAR)
		{
			int var_num = (int)node_value;
			s_vals[top++] = s_vars[var_num];
			continue;
		}

		// not a leaf, will be function
		unsigned int function, outIdx;
		function = (unsigned int)node_value;
		if constexpr (multiOutput) // value(float32) contains the function(int16_t) and outIndex(int16_t) info will using multiOutput mode
		{
			if (is_outNode)
			{
				OutNodeValue v = *(OutNodeValue*) & node_value;
				function = v.function;
				outIdx = v.outIndex;
			}
		}

		float top_val{};
		if (node_type == NodeType::UFUNC)
		{
			float var1 = s_vals[--top];

			if constexpr (multiOutput){
				right_node = var1;
			}

			if (function == Function::SIN)
			{
				top_val = std::sin(var1);
			}
			else if (function == Function::COS)
			{
				top_val = std::cos(var1);
			}
			else if (function == Function::SINH)
			{
				top_val = std::sinh(var1);
			}
			else if (function == Function::TAN)
			{
				top_val = std::tan(var1);
			}
			else if (function == Function::COSH)
			{
				top_val = std::cosh(var1);
			}
			else if (function == Function::TANH)
			{
				top_val = std::tanh(var1);
			}
			else if (function == Function::LOG)
			{
				if (var1 == 0.0f)
				{
					top_val = -MAX_VAL;
				}
				else
				{
					top_val = std::log(std::abs(var1));
				}
			}
			else if (function == Function::INV)
			{
				if (std::abs(var1) <= DELTA)
				{
					var1 = copy_sign(DELTA, var1);
				}
				top_val = 1.0f / var1;
			}
			else if (function == Function::EXP)
			{
				top_val = std::exp(var1);
			}
			else if (function == Function::NEG)
			{
				top_val = -var1;
			}
			else if (function == Function::ABS)
			{
				top_val = std::abs(var1);
			}
			else if (function == Function::SQRT)
			{
				if (var1 <= 0.0f)
				{
					var1 = std::abs(var1);
				}
				top_val = std::sqrt(var1);
			}
		}
		else if (node_type == NodeType::BFUNC)
		{
			float var1 = s_vals[--top];
			float var2 = s_vals[--top];

			if constexpr (multiOutput){
				right_node = var2;
			}

			if (function == Function::ADD)
			{
				top_val = var1 + var2;
			}
			else if (function == Function::SUB)
			{
				top_val = var1 - var2;
			}
			else if (function == Function::MUL)
			{
				top_val = var1 * var2;
			}
			else if (function == Function::DIV)
			{
				if (std::abs(var2) <= DELTA)
				{
					var2 = copy_sign(DELTA, var2);
				}
				top_val = var1 / var2;
			}
			else if (function == Function::POW)
			{
				if (var1 == 0.0f && var2 == 0.0f)
				{
					top_val = 0.0f;
				}
				else
				{
					top_val = std::pow(std::abs(var1), var2);
				}
			}
			else if (function == Function::MAX)
			{
				top_val = var1 >= var2 ? var1 : var2;
			}
			else if (function == Function::MIN)
			{
				top_val = var1 <= var2 ? var1 : var2;
			}
			else if (function == Function::LT)
			{
				top_val = var1 < var2 ? 1 : -1;
			}
			else if (function == Function::GT)
			{
				top_val = var1 > var2 ? 1 : -1;
			}
			else if (function == Function::LE)
			{
				top_val = var1 <= var2 ? 1 : -1;
			}
			else if (function == Function::GE)
			{
				top_val = var1 >= var2 ? 1 : -1;
			}
		}
		else //// if (node_type == NodeType::TFUNC)
		{
			float var1 = s_vals[--top];
			float var2 = s_vals[--top];
			float var3 = s_vals[--top];
			if constexpr (multiOutput){
				right_node = var3;
			}
			//// if (function == Function::IF)
			top_val = var1 > (0.0f) ? var2 : var3;
		}

		// clip value
		if (is_nan(top_val))
		{
			top_val = .0f;
		}
		else if (is_inf(top_val) || std::abs(top_val) > MAX_VAL)
		{	
			top_val = copy_sign(MAX_VAL, top_val);
		}

		// multiple output
		if constexpr (multiOutput)
		{	
			// Y. Zhang and M. Zhang, “A multiple-output program tree structure ingenetic programming,” in Proceedings of. Citeseer, 2004
			if (is_outNode && outIdx < outLen)
				s_outs[outIdx] += top_val;
			top_val = right_node;  // pass right_value to its father
		}

		s_vals[top++] = top_val;
	}
	
	if (top != 1)
	{
		printf("top: %d\n", top);
	}
	assert (top == 1);  // my personal guess
}


template<bool multiOutput = false>
__global__ void treeGPEvalKernel(
    const unsigned int popSize, 
    const unsigned int maxGPLen, 
    const unsigned int varLen, 
    const unsigned int outLen, 
    const float* value,
    const int16_t* type,
    const int16_t* subtree_size,
    const float* variables, 
    float* results
)
{
	const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n >= popSize)
		return;
	if constexpr (multiOutput)
	{
		assert(outLen > 0);
		assert(varLen <= MAX_STACK / 4);
		assert(outLen <= MAX_STACK / 4);  // variable and outputs will load into infos
	}
	else
	{
		assert(varLen <= MAX_STACK / 2);  // varible will load into infos
	}
	// init
	float* stack = (float*)alloca(MAX_STACK * sizeof(float)); // the stack to store the operants
	int16_t* infos = (int16_t*)alloca(2 * MAX_STACK * sizeof(int16_t)); // extra stack memory to load some info

	// current tree
    auto i_value = value + n * maxGPLen; 
    auto i_type = type + n * maxGPLen;
    auto i_subtree_size = subtree_size + n * maxGPLen;

	// current variables
	auto i_vars = variables + n * varLen;

	// call
	float* s_outs{};  // output ptr. default is null. only used in multiOutput mode
	int top{};  // stack ptr.
	_treeGPEvalByStack<multiOutput>(i_value, i_type, i_subtree_size, i_vars, stack, infos, n, popSize, maxGPLen, varLen, outLen, s_outs, top);
	// final
	if constexpr (multiOutput)
	{	
		// load s_outs in results
		auto o_res = results + n * outLen;
		for (int i = 0; i < outLen; i++)
		{
			o_res[i] = s_outs[i];
		}
	}
	else
	{
		results[n] = stack[--top];  // --top will always be 0? I guess
	}
}

void evaluate(
    const unsigned int popSize, 
    const unsigned int maxGPLen, 
    const unsigned int varLen, 
    const unsigned int outLen, 
    const float* value,
    const int16_t* type,
    const int16_t* subtree_size,
    const float* variables, 
    float* results
)
{
	int gridSize{}, blockSize{};
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, treeGPEvalKernel<false>);
	if (gridSize * blockSize < popSize)
		gridSize = (popSize - 1) / blockSize + 1;
	if (outLen > 1)
		treeGPEvalKernel<true><<<gridSize, blockSize>>>(popSize, maxGPLen, varLen, outLen, value, type, subtree_size, variables, results);
	else
		treeGPEvalKernel<false><<<gridSize, blockSize>>>(popSize, maxGPLen, varLen, 0, value, type, subtree_size, variables, results);
}

constexpr auto SR_BLOCK_SIZE = 1024;

template<bool multiOutput = false, bool useMSE = true>
__global__ void treeGPRegressionFitnessKernel(
	const float* value, 
	const int16_t* type, 
	const int16_t* subtree_size, 
	const float* variables, 
	const float* labels, 
	float* fitnesses, 
	const unsigned int popSize, 
	const unsigned int dataPoints, 
	const unsigned int maxGPLen, 
	const unsigned int varLen, 
	const unsigned int outLen = 0
)
/**
 * gps: [popSize * maxLen]
*/
{
	const unsigned int maxThreadBlocks = (dataPoints - 1) / SR_BLOCK_SIZE + 1;
	const unsigned int nGP = blockIdx.x, nTB = blockIdx.y, threadId = threadIdx.x;
	const unsigned int dataPointId = nTB * SR_BLOCK_SIZE + threadId;

	__shared__ float sharedFitness[SR_BLOCK_SIZE];
	sharedFitness[threadId] = .0f;

	if (nGP >= popSize || nTB >= maxThreadBlocks)
		return;
	if constexpr (multiOutput)
	{
		assert(outLen > 0);
		assert(varLen * sizeof(float) / sizeof(int) <= MAX_STACK / 4);
		assert(outLen * sizeof(float) / sizeof(int) <= MAX_STACK / 4);
	}
	else
	{
		assert(varLen * sizeof(float) / sizeof(int) <= MAX_STACK / 2);
	}
	// init

	float fit = .0f;
	float* stack = (float*)alloca(MAX_STACK * sizeof(float));
	int16_t* infos = (int16_t*)alloca(2 * MAX_STACK * sizeof(int16_t));
	
	//current tree
    auto i_value = value + nGP * maxGPLen; 
    auto i_type = type + nGP * maxGPLen;
    auto i_subtree_size = subtree_size + nGP * maxGPLen;

	// evaluate over data points
	if (dataPointId < dataPoints)
	{
		// eval
		auto i_vars = variables + dataPointId * varLen;
		float* s_outs{};
		int top{};
		_treeGPEvalByStack<multiOutput>(i_value, i_type, i_subtree_size, i_vars, stack, infos, nGP, popSize, maxGPLen, varLen, outLen, s_outs, top);
		// accumulate
		if constexpr (multiOutput)
		{
			auto i_labels = labels + dataPointId * outLen;
			for (int i = 0; i < outLen; i++)
			{
				float diff = i_labels[i] - s_outs[i];
				if constexpr (useMSE)
					fit += diff * diff;
				else
					fit += std::abs(diff);  // abs
			}
		}
		else
		{
			float output_value = stack[--top];
			float diff = labels[dataPointId] - output_value;
			if constexpr (useMSE)
				fit = diff * diff;
			else
				fit = std::abs(diff);
			// printf("thread_id: %d, nGP: %d, input: %f, datapoints: %d, dataPointId: %d, fit: %f, labels[dataPointId]: %f, output_value: %f, diff: %f\n", threadId, nGP, i_vars[0], dataPoints, dataPointId, fit, labels[dataPointId], output_value, diff);
		}
	}
	sharedFitness[threadId] = fit;

	__syncthreads();

    for (unsigned int size = SR_BLOCK_SIZE / 2; size > 0; size >>= 1)
    {
        if (threadId < size)
        {
            sharedFitness[threadId] += sharedFitness[threadId + size];
        }
        __syncthreads();
    }

    // 每个block只进行一次atomicAdd
    if (threadId == 0)
    {
        atomicAdd(&fitnesses[nGP], sharedFitness[0]);
    }
}


__global__ void averageFitnessValueKernel(float* fitnesses, const unsigned int popSize, const unsigned int dataPoints){
	const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n >= popSize)
		return;
	fitnesses[n] /= dataPoints;
}


void SR_fitness(
	const unsigned int popSize,
	const unsigned int dataPoints,
	const unsigned int gpLen,
	const unsigned int varLen,
	const unsigned int outLen,
	const bool useMSE,
	const float* value,
	const int16_t* type,
	const int16_t* subtree_size,
	const float* variables, 
	const float* labels, 
	float* fitnesses
)
{
	const unsigned int threadBlocks = (dataPoints - 1) / SR_BLOCK_SIZE + 1;  // number of blocks for one individual
	dim3 gridSize{popSize, threadBlocks};  // total blocks
	auto err = hipMemsetAsync(fitnesses, 0, popSize * sizeof(float));  // clear fitnesses
	if (outLen > 1)
	{
		if (useMSE)
			treeGPRegressionFitnessKernel<true, true><<<gridSize, SR_BLOCK_SIZE>>>(value, type, subtree_size, variables, labels, fitnesses, popSize, dataPoints, gpLen, varLen, outLen);   
		else
			treeGPRegressionFitnessKernel<true, false><<<gridSize, SR_BLOCK_SIZE>>>(value, type, subtree_size, variables, labels, fitnesses, popSize, dataPoints, gpLen, varLen, outLen); 
	}
	else
	{
		if (useMSE)
			treeGPRegressionFitnessKernel<false, true><<<gridSize, SR_BLOCK_SIZE>>>(value, type, subtree_size, variables, labels, fitnesses, popSize, dataPoints, gpLen, varLen, 0); 
		else
			treeGPRegressionFitnessKernel<false, false><<<gridSize, SR_BLOCK_SIZE>>>(value, type, subtree_size, variables, labels, fitnesses, popSize, dataPoints, gpLen, varLen, 0); 
	}

	// average fitness value
	unsigned int averagethreadBlocks = (popSize - 1) / SR_BLOCK_SIZE + 1;
	averageFitnessValueKernel<<<averagethreadBlocks, SR_BLOCK_SIZE>>>(fitnesses, popSize, dataPoints);
}

